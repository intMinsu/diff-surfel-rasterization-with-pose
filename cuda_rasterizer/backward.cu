#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include "math.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

template <typename T>
__device__ void inline reduce_helper(int lane, int i, T *data) {
  if (lane < i) {
    data[lane] += data[lane + i];
  }
}

template <typename group_t, typename... Lists>
__device__ void block_reduction(group_t g, Lists... lists) {
  int lane = g.thread_rank();
  g.sync();

  for (int i = g.size() / 2; i > 0; i /= 2) {
    (...,
     reduce_helper(
         lane, i, lists));
    g.sync();
  }
}

// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ normal_opacity,
	const float* __restrict__ transMats,
	const float* __restrict__ colors,
	const float* __restrict__ depths,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	const float* __restrict__ dL_depths,
	float * __restrict__ dL_dtransMat,
	float3* __restrict__ dL_dmean2D,
	float* __restrict__ dL_dnormal3D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	auto tid = block.thread_rank();
	
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = {(float)pix.x, (float)pix.y};

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];

    __shared__ float2 dL_dmean2D_shared[BLOCK_SIZE];
    __shared__ float3 dL_dcolors_shared[BLOCK_SIZE];
    __shared__ float3 dL_dtransMat_shared_1[BLOCK_SIZE];
	__shared__ float3 dL_dtransMat_shared_2[BLOCK_SIZE];
	__shared__ float3 dL_dtransMat_shared_3[BLOCK_SIZE];	
	__shared__ float dL_dopacity_shared[BLOCK_SIZE];

#if RENDER_AXUTILITY
    __shared__ float3 dL_dnormal3D_shared[BLOCK_SIZE];
#endif

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0.f;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0.f;

	float accum_rec[C] = { 0.f };
	float dL_dpixel[C] = { 0.f };

#if RENDER_AXUTILITY
	float dL_dreg = 0.f;
	float dL_ddepth = 0.f;
	float dL_daccum = 0.f;
	float dL_dnormal2D[3]  = { 0.f };
	const int median_contributor = inside ? n_contrib[pix_id + H * W] : 0.f;
	float dL_dmedian_depth = 0.f;
	float dL_dmax_dweight = 0.f;

	if (inside) {
		dL_ddepth = dL_depths[DEPTH_OFFSET * H * W + pix_id];
		dL_daccum = dL_depths[ALPHA_OFFSET * H * W + pix_id];
		dL_dreg = dL_depths[DISTORTION_OFFSET * H * W + pix_id];
		#pragma unroll
		for (int i = 0; i < 3; i++) 
			dL_dnormal2D[i] = dL_depths[(NORMAL_OFFSET + i) * H * W + pix_id];

		dL_dmedian_depth = dL_depths[MIDDEPTH_OFFSET * H * W + pix_id];
		// dL_dmax_dweight = dL_depths[MEDIAN_WEIGHT_OFFSET * H * W + pix_id];
	}

	// for compute gradient with respect to depth and normal
	float last_depth = 0.f;
	float last_normal[3] = { 0.f };
	float accum_depth_rec = 0.f;
	float accum_alpha_rec = 0.f;
	float accum_normal_rec[3] = {0.f};
	// for compute gradient with respect to the distortion map
	const float final_D = inside ? final_Ts[pix_id + H * W] : 0.f;
	const float final_D2 = inside ? final_Ts[pix_id + 2 * H * W] : 0.f;
	const float final_A = 1 - T_final;
	float last_dL_dT = 0.f;
#endif

	if (inside){
		#pragma unroll
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
	}

	float last_alpha = 0.f;
	float last_color[C] = { 0.f };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5f * W;
	const float ddely_dy = 0.5f * H;
	__shared__ int skip_counter;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		// block.sync();
		const int progress = i * BLOCK_SIZE + tid;
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[tid] = coll_id;
			collected_xy[tid] = points_xy_image[coll_id];
			collected_normal_opacity[tid] = normal_opacity[coll_id];
			collected_Tu[tid] = {transMats[9 * coll_id+0], transMats[9 * coll_id+1], transMats[9 * coll_id+2]};
			collected_Tv[tid] = {transMats[9 * coll_id+3], transMats[9 * coll_id+4], transMats[9 * coll_id+5]};
			collected_Tw[tid] = {transMats[9 * coll_id+6], transMats[9 * coll_id+7], transMats[9 * coll_id+8]};

			#pragma unroll
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + tid] = colors[coll_id * C + i];
		}
		// block.sync();

		// Iterate over Gaussians
		for (int j = 0; j < min(BLOCK_SIZE, toDo); j++)
		{

			block.sync();
			if (tid == 0) {
				skip_counter = 0;
			}
			block.sync();

			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			// contributor--;
			// if (contributor >= last_contributor)
			// 	continue;

			bool skip = done;
			contributor = done ? contributor : contributor - 1;
			skip |= contributor >= last_contributor;

			// compute ray-splat intersection as before
			// Fisrt compute two homogeneous planes, See Eq. (8)
			const float2 xy = collected_xy[j];
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];
			float3 k = pix.x * Tw - Tu;
			float3 l = pix.y * Tw - Tv;
			float3 p = cross(k, l);
			// if (p.z == 0.0) continue;
			skip |= (p.z == 0);
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); 
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y); 

			// compute intersection and depth
			float rho = min(rho3d, rho2d);
			float c_d = (rho3d <= rho2d) ? (s.x * Tw.x + s.y * Tw.y) + Tw.z : Tw.z; 
			// if (c_d < near_n) continue;
			skip |= (c_d < near_n);
			float4 nor_o = collected_normal_opacity[j];
			float normal[3] = {nor_o.x, nor_o.y, nor_o.z};
			float opa = nor_o.w;

			// accumulations

			float power = -0.5f * rho;
			// if (power > 0.0f)
			// 	continue;

			skip |= (power > 0.0f);

			const float G = exp(power);
			const float alpha = min(0.99f, opa * G);
			// if (alpha < 1.0f / 255.0f)
			// 	continue;

			skip |= (alpha < 1.0f / 255.0f);


			if (skip) {
				atomicAdd(&skip_counter, 1);
			}
			block.sync();

			if (skip_counter == BLOCK_SIZE) {
				continue;
			}

			T = skip ? T : T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;
			const float w = alpha * T;
			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			float local_dL_dcolors[3];

			#pragma unroll
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = skip ? accum_rec[ch] : last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = skip ? last_color[ch] : c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				// atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
				local_dL_dcolors[ch] = skip ? 0.0f : dchannel_dcolor * dL_dchannel;
			}

			dL_dcolors_shared[tid].x = local_dL_dcolors[0];
			dL_dcolors_shared[tid].y = local_dL_dcolors[1];
			dL_dcolors_shared[tid].z = local_dL_dcolors[2];

			float dL_dz = 0.0f;
			float dL_dweight = 0;


#if RENDER_AXUTILITY
			const float m_d = far_n / (far_n - near_n) * (1 - near_n / c_d);
			const float dmd_dd = (far_n * near_n) / ((far_n - near_n) * c_d * c_d);
			if (contributor == median_contributor-1) {
				dL_dz += dL_dmedian_depth;
				// dL_dweight += dL_dmax_dweight;
			}
#if DETACH_WEIGHT 
			// if not detached weight, sometimes 
			// it will bia toward creating extragated 2D Gaussians near front
			dL_dweight += 0;
#else
			dL_dweight += (final_D2 + m_d * m_d * final_A - 2 * m_d * final_D) * dL_dreg;
#endif
			dL_dalpha += dL_dweight - last_dL_dT;
			// propagate the current weight W_{i} to next weight W_{i-1}
			last_dL_dT = skip ? last_dL_dT : dL_dweight * alpha + (1 - alpha) * last_dL_dT;
			const float dL_dmd = 2.0f * (T * alpha) * (m_d * final_A - final_D) * dL_dreg;
			dL_dz += dL_dmd * dmd_dd;

			// Propagate gradients w.r.t ray-splat depths
			accum_depth_rec = skip ? accum_depth_rec : last_alpha * last_depth + (1.f - last_alpha) * accum_depth_rec;
			last_depth = skip? last_depth : c_d;
			dL_dalpha += (c_d - accum_depth_rec) * dL_ddepth;
			// Propagate gradients w.r.t. color ray-splat alphas
			accum_alpha_rec = skip ? accum_alpha_rec : last_alpha * 1.0 + (1.f - last_alpha) * accum_alpha_rec;
			dL_dalpha += (1 - accum_alpha_rec) * dL_daccum;

			// Propagate gradients to per-Gaussian normals

			float local_dL_dnormal2D[3];
			#pragma unroll
			for (int ch = 0; ch < 3; ch++) {
				// accum_normal_rec[ch] = last_alpha * last_normal[ch] + (1.f - last_alpha) * accum_normal_rec[ch];
				// last_normal[ch] = normal[ch];
				accum_normal_rec[ch] = skip ? accum_normal_rec[ch] : last_alpha * last_normal[ch] + (1.f - last_alpha) * accum_normal_rec[ch];
				last_normal[ch] = skip ? last_normal[ch] : normal[ch];

				dL_dalpha += (normal[ch] - accum_normal_rec[ch]) * dL_dnormal2D[ch];
				local_dL_dnormal2D[ch] = skip ? 0.0f : dL_dnormal2D[ch];
				// atomicAdd((&dL_dnormal3D[global_id * 3 + ch]), alpha * T * dL_dnormal2D[ch]);
			}

			dL_dnormal3D_shared[tid] = make_float3(local_dL_dnormal2D[0], local_dL_dnormal2D[1], local_dL_dnormal2D[2]);
#endif

			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			// last_alpha = alpha;
			last_alpha = skip ? last_alpha : alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0.f;
			#pragma unroll
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			const float dL_dG = nor_o.w * dL_dalpha;
#if RENDER_AXUTILITY
			dL_dz += alpha * T * dL_ddepth; 
#endif

			if (rho3d <= rho2d) {
				// Update gradients w.r.t. covariance of Gaussian 3x3 (T)
				const float2 dL_ds = {
					dL_dG * -G * s.x + dL_dz * Tw.x,
					dL_dG * -G * s.y + dL_dz * Tw.y
				};
				const float3 dz_dTw = {s.x, s.y, 1.0};
				const float dsx_pz = dL_ds.x / p.z;
				const float dsy_pz = dL_ds.y / p.z;
				const float3 dL_dp = {dsx_pz, dsy_pz, -(dsx_pz * s.x + dsy_pz * s.y)};
				const float3 dL_dk = cross(l, dL_dp);
				const float3 dL_dl = cross(dL_dp, k);

				const float3 dL_dTu = {-dL_dk.x, -dL_dk.y, -dL_dk.z};
				const float3 dL_dTv = {-dL_dl.x, -dL_dl.y, -dL_dl.z};
				const float3 dL_dTw = {
					pixf.x * dL_dk.x + pixf.y * dL_dl.x + dL_dz * dz_dTw.x, 
					pixf.x * dL_dk.y + pixf.y * dL_dl.y + dL_dz * dz_dTw.y, 
					pixf.x * dL_dk.z + pixf.y * dL_dl.z + dL_dz * dz_dTw.z};

				dL_dtransMat_shared_1[tid] = skip ? make_float3(0.f, 0.f, 0.f) : dL_dTu;
				dL_dtransMat_shared_2[tid] = skip ? make_float3(0.f, 0.f, 0.f) : dL_dTv;
				dL_dtransMat_shared_3[tid] = skip ? make_float3(0.f, 0.f, 0.f) : dL_dTw;
				dL_dmean2D_shared[tid] = make_float2(0.f, 0.f);

			} else {
				// // Update gradients w.r.t. center of Gaussian 2D mean position
				const float dG_ddelx = -G * FilterInvSquare * d.x;
				const float dG_ddely = -G * FilterInvSquare * d.y;


				dL_dmean2D_shared[tid].x = skip ? 0.f : dL_dG * dG_ddelx;
				dL_dmean2D_shared[tid].y = skip ? 0.f : dL_dG * dG_ddely;

				dL_dtransMat_shared_1[tid] = make_float3(0.f, 0.f, 0.f);
				dL_dtransMat_shared_2[tid] = make_float3(0.f, 0.f, 0.f);
				dL_dtransMat_shared_3[tid] = skip ? make_float3(0.f, 0.f, 0.f) : make_float3(s.x * dL_dz, s.y * dL_dz, dL_dz);

			}

			// Update gradients w.r.t. opacity of the Gaussian
			dL_dopacity_shared[tid] = skip ? 0.f :  G * dL_dalpha;

			block_reduction(block, dL_dcolors_shared, dL_dopacity_shared, 
			dL_dtransMat_shared_1,dL_dtransMat_shared_2, dL_dtransMat_shared_3,
			dL_dmean2D_shared, dL_dnormal3D_shared);


			if (tid == 0) {
				float2 dL_dmean2D_acc = dL_dmean2D_shared[0];
				float dL_dopacity_acc = dL_dopacity_shared[0];
				float3 dL_dcolors_acc = dL_dcolors_shared[0];
				float3 dL_dnormal3D_acc = dL_dnormal3D_shared[0];
				float3 dL_dtransMat_acc_1 = dL_dtransMat_shared_1[0];
				float3 dL_dtransMat_acc_2 = dL_dtransMat_shared_2[0];
				float3 dL_dtransMat_acc_3 = dL_dtransMat_shared_3[0];

				atomicAdd(&dL_dmean2D[global_id].x, dL_dmean2D_acc.x);
				atomicAdd(&dL_dmean2D[global_id].y, dL_dmean2D_acc.y);
				atomicAdd(&dL_dopacity[global_id], dL_dopacity_acc);

				atomicAdd(&dL_dcolors[global_id * 3 + 0], dL_dcolors_acc.x);
				atomicAdd(&dL_dcolors[global_id * 3 + 1], dL_dcolors_acc.y);
				atomicAdd(&dL_dcolors[global_id * 3 + 2], dL_dcolors_acc.z);

				atomicAdd(&dL_dnormal3D[global_id * 3 + 0], dL_dnormal3D_acc.x);
				atomicAdd(&dL_dnormal3D[global_id * 3 + 1], dL_dnormal3D_acc.y);
				atomicAdd(&dL_dnormal3D[global_id * 3 + 2], dL_dnormal3D_acc.z);

				atomicAdd(&dL_dtransMat[global_id * 9 + 0], dL_dtransMat_acc_1.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 1], dL_dtransMat_acc_1.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 2], dL_dtransMat_acc_1.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 3], dL_dtransMat_acc_2.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 4], dL_dtransMat_acc_2.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 5], dL_dtransMat_acc_2.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 6], dL_dtransMat_acc_3.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 7], dL_dtransMat_acc_3.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 8], dL_dtransMat_acc_3.z);

			}

		}
	}
}


__device__ void compute_transmat_aabb(
	int idx, 
	const float* Ts_precomp,
	const float3* p_origs, 
	const glm::vec2* scales, 
	const glm::vec4* rots, 
	const float* projmatrix, 
	const float* projmatrix_raw,
	const float* viewmatrix, 
	const int W, const int H, 
	const float3* dL_dnormals,
	const float3* dL_dmean2Ds, 
	float* dL_dTs, 
	glm::vec3* dL_dmeans, 
	glm::vec2* dL_dscales,
	 glm::vec4* dL_drots,
	 float* dL_dtau)
{
	glm::mat3 T;
	float3 normal;
	glm::mat3x4 P;
	glm::mat3 R;
	glm::mat3 S;
	glm::mat3 L;
	float3 p_orig;
	glm::vec4 rot;
	glm::vec2 scale;
	glm::mat3x4 M;
	glm::mat4x3 Mt;
	glm::mat3x4 view2pix;
	glm::mat4 viewmat_glm;
	// Get transformation matrix of the Gaussian
	if (Ts_precomp != nullptr) {
		T = glm::mat3(
			Ts_precomp[idx * 9 + 0], Ts_precomp[idx * 9 + 1], Ts_precomp[idx * 9 + 2],
			Ts_precomp[idx * 9 + 3], Ts_precomp[idx * 9 + 4], Ts_precomp[idx * 9 + 5],
			Ts_precomp[idx * 9 + 6], Ts_precomp[idx * 9 + 7], Ts_precomp[idx * 9 + 8]
		);
		normal = {0.0, 0.0, 0.0};
	} else {
		p_orig = p_origs[idx];
		rot = rots[idx];
		scale = scales[idx];
		R = quat_to_rotmat(rot);
		S = scale_to_mat(scale, 1.0f);
		
		L = R * S;

		M = glm::mat3x4(
			glm::vec4(L[0], 0.0),
			glm::vec4(L[1], 0.0),
			glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
		);
		Mt = glm::transpose(M);

		viewmat_glm = glm::mat4(
			viewmatrix[0], viewmatrix[4], viewmatrix[8], viewmatrix[12],
			viewmatrix[1], viewmatrix[5], viewmatrix[9], viewmatrix[13],
			viewmatrix[2], viewmatrix[6], viewmatrix[10], viewmatrix[14],
			viewmatrix[3], viewmatrix[7], viewmatrix[11], viewmatrix[15]
		);

		glm::mat4 projmat_glm = glm::mat4(
			projmatrix_raw[0], projmatrix_raw[4], projmatrix_raw[8], projmatrix_raw[12],
			projmatrix_raw[1], projmatrix_raw[5], projmatrix_raw[9], projmatrix_raw[13],
			projmatrix_raw[2], projmatrix_raw[6], projmatrix_raw[10], projmatrix_raw[14],
			projmatrix_raw[3], projmatrix_raw[7], projmatrix_raw[11], projmatrix_raw[15]
		);
		

		glm::mat3x4 ndc2pix = glm::mat3x4(
			glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
			glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
			glm::vec4(0.0, 0.0, 0.0, 1.0)
		);

		view2pix = projmat_glm * ndc2pix;

		

		P = viewmat_glm * view2pix;
		T = Mt * P;

		normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);
	}

	// Update gradients w.r.t. transformation matrix of the Gaussian
	glm::mat3 dL_dT = glm::mat3(
		dL_dTs[idx*9+0], dL_dTs[idx*9+1], dL_dTs[idx*9+2],
		dL_dTs[idx*9+3], dL_dTs[idx*9+4], dL_dTs[idx*9+5],
		dL_dTs[idx*9+6], dL_dTs[idx*9+7], dL_dTs[idx*9+8]
	);


	float3 dL_dmean2D = dL_dmean2Ds[idx];
	if(dL_dmean2D.x != 0 || dL_dmean2D.y != 0)
	{
		glm::vec3 t_vec = glm::vec3(9.0f, 9.0f, -1.0f);
		float d = glm::dot(t_vec, T[2] * T[2]);
		glm::vec3 f_vec = t_vec * (1.0f / d);
		glm::vec3 dL_dT0 = dL_dmean2D.x * f_vec * T[2];
		glm::vec3 dL_dT1 = dL_dmean2D.y * f_vec * T[2];
		glm::vec3 dL_dT3 = dL_dmean2D.x * f_vec * T[0] + dL_dmean2D.y * f_vec * T[1];
		glm::vec3 dL_df = dL_dmean2D.x * T[0] * T[2] + dL_dmean2D.y * T[1] * T[2];
		float dL_dd = glm::dot(dL_df, f_vec) * (-1.0 / d);
		glm::vec3 dd_dT3 = t_vec * T[2] * 2.0f;
		dL_dT3 += dL_dd * dd_dT3;
		dL_dT[0] += dL_dT0;
		dL_dT[1] += dL_dT1;
		dL_dT[2] += dL_dT3;

		if (Ts_precomp != nullptr) {
			dL_dTs[idx * 9 + 0] = dL_dT[0].x;
			dL_dTs[idx * 9 + 1] = dL_dT[0].y;
			dL_dTs[idx * 9 + 2] = dL_dT[0].z;
			dL_dTs[idx * 9 + 3] = dL_dT[1].x;
			dL_dTs[idx * 9 + 4] = dL_dT[1].y;
			dL_dTs[idx * 9 + 5] = dL_dT[1].z;
			dL_dTs[idx * 9 + 6] = dL_dT[2].x;
			dL_dTs[idx * 9 + 7] = dL_dT[2].y;
			dL_dTs[idx * 9 + 8] = dL_dT[2].z;
			return;
		}
	}
	
	if (Ts_precomp != nullptr) return;


	glm::mat3x4 dL_dM = P * glm::transpose(dL_dT);
	float3 dL_dtn = transformVec4x3Transpose(dL_dnormals[idx], viewmatrix);

	

	float dL_dT_data[9];
	dL_dT_data[0] = dL_dTs[idx*9+0]; dL_dT_data[3] = dL_dTs[idx*9+1]; dL_dT_data[6] = dL_dTs[idx*9+2];
	dL_dT_data[1] = dL_dTs[idx*9+3]; dL_dT_data[4] = dL_dTs[idx*9+4]; dL_dT_data[7] = dL_dTs[idx*9+5];
	dL_dT_data[2] = dL_dTs[idx*9+6]; dL_dT_data[5] = dL_dTs[idx*9+7]; dL_dT_data[8] = dL_dTs[idx*9+8];


	mat33 dL_dT_3x3 = mat33(
			dL_dT_data
	);
	float Mt_data[9] = {Mt[0][0], Mt[1][0], Mt[2][0],
		Mt[0][1], Mt[1][1], Mt[2][1],
		Mt[0][2], Mt[1][2], Mt[2][2]};

	mat33 Mt_3x3 = mat33(
		Mt_data
	);

	mat33 dL_dT_3x3_t = dL_dT_3x3.transpose();
	mat33 M_3x3 = Mt_3x3.transpose();

	glm::mat3x4 dL_dP_glm = M * dL_dT;
	
	float dL_dViewMat_00 = dL_dP_glm[0][0] * view2pix[0][0]; float dL_dViewMat_01 = dL_dP_glm[0][1] * view2pix[0][0]; float dL_dViewMat_02 = dL_dP_glm[0][2] * view2pix[0][0]; float dL_dViewMat_03 = dL_dP_glm[0][3] * view2pix[0][0];
	float dL_dViewMat_10 = dL_dP_glm[1][0] * view2pix[1][1]; float dL_dViewMat_11 = dL_dP_glm[1][1] * view2pix[1][1]; float dL_dViewMat_12 = dL_dP_glm[1][2] * view2pix[1][1]; float dL_dViewMat_13 = dL_dP_glm[1][3] * view2pix[1][1];
	
	float dL_dViewMat_20 = dL_dP_glm[0][0] * view2pix[0][2] + dL_dP_glm[1][0] * view2pix[1][2] + dL_dP_glm[2][0];
	float dL_dViewMat_21 = dL_dP_glm[0][1] * view2pix[0][2] + dL_dP_glm[1][1] * view2pix[1][2] + dL_dP_glm[2][1]; 
	float dL_dViewMat_22 = dL_dP_glm[0][2] * view2pix[0][2] + dL_dP_glm[1][2] * view2pix[1][2] + dL_dP_glm[2][2]; 
	float dL_dViewMat_23 = dL_dP_glm[0][3] * view2pix[0][2] + dL_dP_glm[1][3] * view2pix[1][2] + dL_dP_glm[2][3];


	dL_dViewMat_00 += dL_dnormals[idx].x *L[2].x; dL_dViewMat_01 += dL_dnormals[idx].x *L[2].y; dL_dViewMat_02 += dL_dnormals[idx].x *L[2].z;
	dL_dViewMat_10 += dL_dnormals[idx].y *L[2].x; dL_dViewMat_11 += dL_dnormals[idx].y *L[2].y; dL_dViewMat_12 += dL_dnormals[idx].y *L[2].z;
	dL_dViewMat_20 += dL_dnormals[idx].z *L[2].x; dL_dViewMat_21 += dL_dnormals[idx].z *L[2].y; dL_dViewMat_22 += dL_dnormals[idx].z *L[2].z;


	SE3 T_CW(viewmatrix);
	float3 t_ = T_CW.t();
	mat33 Rot = T_CW.R().data();

	float3 c1 = Rot.cols[0];
	float3 c2 = Rot.cols[1];
	float3 c3 = Rot.cols[2];
	mat33 skew1 = - mat33::skew_symmetric(c1);
	mat33 skew2 = - mat33::skew_symmetric(c2);
	mat33 skew3 = - mat33::skew_symmetric(c3);
	mat33 skewt = - mat33::skew_symmetric(t_);

	float dL_dView_data[12];
	dL_dView_data[0] = dL_dViewMat_00; dL_dView_data[3] = dL_dViewMat_01; dL_dView_data[6] = dL_dViewMat_02; dL_dView_data[9] = dL_dViewMat_03; 
	dL_dView_data[1] = dL_dViewMat_10; dL_dView_data[4] = dL_dViewMat_11; dL_dView_data[7] = dL_dViewMat_12; dL_dView_data[10] = dL_dViewMat_13; 
	dL_dView_data[2] = dL_dViewMat_20; dL_dView_data[5] = dL_dViewMat_21; dL_dView_data[8] = dL_dViewMat_22; dL_dView_data[11] = dL_dViewMat_23;
	mat34 dL_dView(dL_dView_data);

	float3 c1_view = dL_dView.cols[0]; float3 c2_view = dL_dView.cols[1]; 
	float3 c3_view = dL_dView.cols[2]; float3 t_view = dL_dView.cols[3];

	float3 dL_drho;
	dL_drho.x = dL_dViewMat_03; dL_drho.y = dL_dViewMat_13; dL_drho.z = dL_dViewMat_23;


	float3 dL_dtheta;
	dL_dtheta.x = dot(c1_view, skew1.cols[0]) + dot(c2_view, skew2.cols[0]) + dot(c3_view, skew3.cols[0]) + dot(t_view, skewt.cols[0]);
	dL_dtheta.y = dot(c1_view, skew1.cols[1]) + dot(c2_view, skew2.cols[1]) + dot(c3_view, skew3.cols[1]) + dot(t_view, skewt.cols[1]);
	dL_dtheta.z = dot(c1_view, skew1.cols[2]) + dot(c2_view, skew2.cols[2]) + dot(c3_view, skew3.cols[2]) + dot(t_view, skewt.cols[2]);

	float dL_dt[6] = {dL_drho.x, dL_drho.y, dL_drho.z,
	 dL_dtheta.x, dL_dtheta.y, dL_dtheta.z};

	for(int i=0; i<6; i++){
		dL_dtau[6*idx+i] += dL_dt[i];
	}

#if DUAL_VISIABLE
	float3 p_view = transformPoint4x3(p_orig, viewmatrix);
	float cos = -sumf3(p_view * normal);
	float multiplier = cos > 0 ? 1: -1;
	dL_dtn = multiplier * dL_dtn;
#endif
	glm::mat3 dL_dRS = glm::mat3(
		glm::vec3(dL_dM[0]),
		glm::vec3(dL_dM[1]),
		glm::vec3(dL_dtn.x, dL_dtn.y, dL_dtn.z)
	);

	glm::mat3 dL_dR = glm::mat3(
		dL_dRS[0] * glm::vec3(scale.x),
		dL_dRS[1] * glm::vec3(scale.y),
		dL_dRS[2]);
	
	dL_drots[idx] = quat_to_rotmat_vjp(rot, dL_dR);
	dL_dscales[idx] = glm::vec2(
		(float)glm::dot(dL_dRS[0], R[0]),
		(float)glm::dot(dL_dRS[1], R[1])
	);
	dL_dmeans[idx] = glm::vec3(dL_dM[2]);
}

template<int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3* means3D,
	const float* transMats,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
	const float* proj_raw,
	const float focal_x, 
	const float focal_y,
	const float tan_fovx,
	const float tan_fovy,
	const glm::vec3* campos, 
	// grad input
	float* dL_dtransMats,
	const float* dL_dnormal3Ds,
	float* dL_dcolors,
	float* dL_dshs,
	float3* dL_dmean2Ds,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots,
	float* dL_dtau)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	const int W = int(focal_x * tan_fovx * 2);
	const int H = int(focal_y * tan_fovy * 2);
	const float * Ts_precomp = (scales) ? nullptr : transMats;
	compute_transmat_aabb(
		idx, 
		Ts_precomp,
		means3D, scales, rotations, 
		projmatrix, proj_raw, viewmatrix, W, H, 
		(float3*)dL_dnormal3Ds, 
		dL_dmean2Ds,
		(dL_dtransMats), 
		dL_dmean3Ds, 
		dL_dscales, 
		dL_drots,
		dL_dtau
	);

	if (shs)
		computeColorFromSH(idx, D, M, (glm::vec3*)means3D, *campos, shs, clamped, (glm::vec3*)dL_dcolors, (glm::vec3*)dL_dmean3Ds, (glm::vec3*)dL_dshs);
	
	// hack the gradient here for densitification
	float depth = transMats[idx * 9 + 8];
	dL_dmean2Ds[idx].x = dL_dtransMats[idx * 9 + 2] * depth * 0.5 * float(W); // to ndc 
	dL_dmean2Ds[idx].y = dL_dtransMats[idx * 9 + 5] * depth * 0.5 * float(H); // to ndc
}


void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* transMats,
	const float* viewmatrix,
	const float* projmatrix,
	const float* projmatrix_raw,
    const float focal_x, float focal_y,
	const float cx, float cy,
	const float tan_fovx, const float tan_fovy,
	const glm::vec3* campos, 
	float3* dL_dmean2Ds,
	const float* dL_dnormal3Ds,
	float* dL_dtransMats,
	float* dL_dcolors,
	float* dL_dshs,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots,
	float* dL_dtau)
{	
	preprocessCUDA<NUM_CHANNELS><< <(P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		transMats,
		radii,
		shs,
		clamped,
		(glm::vec2*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		projmatrix_raw,
		focal_x, 
		focal_y,
		tan_fovx,
		tan_fovy,
		campos,	
		dL_dtransMats,
		dL_dnormal3Ds,
		dL_dcolors,
		dL_dshs,
		dL_dmean2Ds,
		dL_dmean3Ds,
		dL_dscales,
		dL_drots,
		dL_dtau
	);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* bg_color,
	const float2* means2D,
	const float4* normal_opacity,
	const float* colors,
	const float* transMats,
	const float* depths,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_depths,
	float * dL_dtransMat,
	float3* dL_dmean2D,
	float* dL_dnormal3D,
	float* dL_dopacity,
	float* dL_dcolors)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		bg_color,
		means2D,
		normal_opacity,
		transMats,
		colors,
		depths,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_depths,
		dL_dtransMat,
		dL_dmean2D,
		dL_dnormal3D,
		dL_dopacity,
		dL_dcolors
		);
}